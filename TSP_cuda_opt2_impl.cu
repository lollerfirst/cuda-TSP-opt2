#include "hip/hip_runtime.h"
#include <time.h>
#include <stdlib.h>
#include <stdio.h>
#include <stdbool.h>
#include <stdint.h>
#include <hip/hip_runtime_api.h>

#define NUM_CITIES 25
#define MAX_DISTANCE 32767
#define MEM_ALIGNMENT 32
#define BLOCK_SIZE 512

#define BUFFER_LEN ((NUM_CITIES * (NUM_CITIES - 1)) / 2)
#define ALIGNED_UNIT_SIZE ((((sizeof(int) * NUM_CITIES + sizeof(int)) / MEM_ALIGNMENT) + 1) * MEM_ALIGNMENT)
#define NUM_OPTS (((NUM_CITIES * (NUM_CITIES - 3)) / 2) + 1)

// Distance matrix represented in compressed form
static int cities[BUFFER_LEN];

// device data
__constant__ int device_cities[BUFFER_LEN];

// indexes the compressed sparse matrix holding the distances
__inline__ __host__ __device__ int triu_index(const int i, const int j)
{
	const int side_i = NUM_CITIES - (i+1);
	const int side_j = NUM_CITIES - (j+1);
	const int sub_value_i = side_i*(side_i+1)/2;
	const int sub_value_j = side_j*(side_j+1)/2;

	return ((BUFFER_LEN - sub_value_i) + j - i - 1 * (i < j)) +
		((BUFFER_LEN - sub_value_j) + i - j - 1 * (j < i));
}

// build the data structure on the host
void build_cities(unsigned int seed)
{
	srand(seed);
	 
	int i;
	for (i=0; i<BUFFER_LEN; ++i)
	{	
		cities[i] = rand() % MAX_DISTANCE;
	}
}

int greedy_path_dist(int* path, int initial_idx)
{
	int distance = 0;
	size_t i;
	int idx = initial_idx;
	
	bool visited_cities[NUM_CITIES] = {0};
	
	// Can't choose the initial before having ended the tour
	visited_cities[initial_idx] = true;
	
	
	// For every node in the path
	for (i=0; i<NUM_CITIES; ++i)
	{		
		if (i != NUM_CITIES - 1)
		{
			int best_dist = INT_MAX;
			int best_idx = -1;
			size_t j;
			
			// For every possible link
			for (j=0; j<NUM_CITIES; ++j)
			{
				if (idx != j &&
					cities[triu_index(idx, j)] <= best_dist &&
					!visited_cities[j])
				{
					best_dist = cities[triu_index(idx, j)];
					best_idx = j;
				}
			}
			
			if (best_idx == -1)
			{
				return -1;
			}
			
			visited_cities[best_idx] = true;
			path[i] = best_idx;
			idx = best_idx;
			distance += best_dist;
		}
		else
		{
			// LAST MUST BE the initial idx
			path[i] = initial_idx;
			distance += cities[triu_index(idx, initial_idx)];
		}
	}
	
	return distance;
}

__inline__ __device__ bool trylock(int* mutex)
{
	// Aquire the lock with an atomic compare exchange
	int old = atomicCAS(mutex, 0, 1);
	
	if (old == 1)
	{
		return false;
	}
	else
	{
		return true;
	}
}

__inline__ __device__ void unlock(int* mutex)
{
	// Release the lock with an atomic exchange
	(void) atomicExch(mutex, 0);
}

template <typename TYPE>
__global__ void copy(TYPE* dest, const TYPE* src, size_t count)
{
	int tid = threadIdx.x + (blockIdx.x * blockDim.x);
	if (tid < count)
	{
		dest[tid] = src[tid];
	}
	
	return;
}

// Worker threads
__global__ void cuda_calculate_opts(int* memory_block,
	bool switched_pointers,
	int initial_idx)
{

	// Thread identification
	int tid = threadIdx.x + (blockIdx.x * blockDim.x);
	
	// Boundaries control
	if (tid >= NUM_OPTS)
	{
		return;
	}
	
	__shared__ int lock;
	lock = 0;
	
	int* current_path = memory_block;
		
	int* output_path = memory_block + (ALIGNED_UNIT_SIZE / sizeof(int)) * (blockIdx.x + 1);
	
	// ENUMERATION OF OPTS:
	int swap_b = 0;
	int accumulator = NUM_CITIES - 2;
	int range = accumulator;
	
	while (tid / accumulator > 0)
	{
		
		++swap_b;
		
		if (swap_b >= NUM_CITIES - 2)
		{
			return;
		}
		
		range = accumulator;
		accumulator += (NUM_CITIES - 2) - swap_b;
	}
	
	// warp-scoped branching causes a mess without it
	__syncwarp();
	

	int swap_bin[2];
	
	
	int swap_a = swap_b + (tid % range) + 1;
	int distance = current_path[NUM_CITIES];
	
	// Load only the nodes to swap
	swap_bin[1] = current_path[swap_b];
	swap_bin[0] = current_path[swap_a];
	
	
	// RECALCULATE DISTANCE:
	// subtract distance from swap_b - 1 to swap_b and from swap_b to swap_b + 1
	// subtract distance from swap_a - 1 to swap_a and from swap_a to swap_a + 1
	// If swap_b + 1 is swap_a and swap_a - 1 is swap_b, subtract 0.
	distance -= (swap_b > 0) ? device_cities[triu_index(current_path[swap_b-1], current_path[swap_b])] : 0;
	
	distance -= device_cities[triu_index(initial_idx, current_path[swap_b])]
		* (swap_b == 0);
	distance -= device_cities[triu_index(current_path[swap_b], current_path[swap_b+1])]
		* (swap_b + 1 != swap_a);
	distance -= device_cities[triu_index(current_path[swap_a-1], current_path[swap_a])]
		* (swap_a - 1 != swap_b);
	distance -= device_cities[triu_index(current_path[swap_a], current_path[swap_a+1])];
	
	// add distance from swap_b - 1 to swap_a and from swap_a to swap_b + 1
	// add distance from swap_a - 1 to swap_b and from swap_b to swap_a + 1
	// If swap_b + 1 is swap_a and swap_a - 1 is swap_b, add 0.
	distance += (swap_b > 0) ? device_cities[triu_index(current_path[swap_b-1], current_path[swap_a])] : 0;
	
	distance += device_cities[triu_index(initial_idx, current_path[swap_a])]
		* (swap_b == 0);
	distance += device_cities[triu_index(current_path[swap_a], current_path[swap_b+1])]
		* (swap_b + 1 != swap_a);
	distance += device_cities[triu_index(current_path[swap_a-1], current_path[swap_b])]
		* (swap_a - 1 != swap_b);
	distance += device_cities[triu_index(current_path[swap_b], current_path[swap_a+1])];
	
	// Block-wide sync
	__syncthreads();

	// Acquire the lock
	while (trylock(&lock) == false);
	
	if (distance < current_path[NUM_CITIES])
	{
		output_path[NUM_CITIES] = distance;
	
		constexpr int num_blocks_copy = (NUM_CITIES + BLOCK_SIZE + 1) / BLOCK_SIZE;
		copy<<<num_blocks_copy, BLOCK_SIZE>>>(output_path, current_path, NUM_CITIES);
		
		output_path[swap_a] = swap_bin[1];
		output_path[swap_b] = swap_bin[0];
	}	
	
	// Release the lock
	unlock(&lock);
	__syncwarp();
	
	return;
}

// Control thread
__global__ void cuda_opt2(int* memory_block, int initial_idx)
{

	constexpr int num_blocks = (NUM_OPTS + BLOCK_SIZE - 1) / BLOCK_SIZE;
	constexpr int num_blocks_copy = ((ALIGNED_UNIT_SIZE/sizeof(int)) + BLOCK_SIZE + 1) / BLOCK_SIZE;
	
	int new_best_dist = memory_block[NUM_CITIES];
	int old_best_dist = new_best_dist + 1;
	bool switched_pointers = false;
	
	int best_index = -1;
	
	while (new_best_dist < old_best_dist) 
	{
		// THIS SWITCH CUTS OFF THE UNNECESSARY COPY KERNEL CALL AT THE BEGINNING
		switch (best_index)
		{
		default:
			// Copy best path and distance to current path
			copy<<<num_blocks_copy, BLOCK_SIZE>>>(
				memory_block,
				&memory_block[(ALIGNED_UNIT_SIZE/sizeof(int)) * best_index],
				ALIGNED_UNIT_SIZE / sizeof(int)
			);
			
			// Wait for child grid to terminate
			hipDeviceSynchronize();
		
		case -1:
			
			// save previous calculated distance
			old_best_dist = new_best_dist;

			// Launch kernel that computes paths and distances
			cuda_calculate_opts<<<num_blocks, BLOCK_SIZE>>>(
				memory_block,
				switched_pointers,
				initial_idx
			);

			// Wait for child grid to terminate
			hipDeviceSynchronize();

			hipError_t err_code = hipGetLastError();
			if (err_code)
			{
				return;
			}

			// retrieve best calculated distance amongst various blocks
			for (int i=1; i<num_blocks+1; ++i)
			{
				int calc_distance = memory_block[(ALIGNED_UNIT_SIZE/sizeof(int)) * i + NUM_CITIES];
				new_best_dist = (calc_distance < new_best_dist) ? calc_distance : new_best_dist;
				best_index = (calc_distance < new_best_dist) ? i : best_index; 
			}
		}
	}

	
	return;
}


int main(void)
{
	// Build the data structure
	build_cities(1);

	// Errors
	hipError_t err_code;

	// Copy cities from host to device, cities is costant
	err_code = hipMemcpyToSymbol(HIP_SYMBOL(device_cities),
		cities,
		sizeof(int) * (BUFFER_LEN)
	);

	if (err_code)
	{
		printf("[!] Cuda Error at line %d: %s\n", __LINE__, hipGetErrorName(err_code));
		return -1;
	}

	// Calculate number of blocks necessary
	constexpr int num_blocks = (NUM_OPTS + BLOCK_SIZE - 1) / BLOCK_SIZE;

	int* memory_block;

	
	// Calculate memory block size
	size_t memory_block_size = ALIGNED_UNIT_SIZE * (num_blocks+1);

	// Allocate memory
	hipMalloc((void**) &memory_block,
	    memory_block_size
	);
	
	if (err_code)
	{
		printf("[!] Cuda Error at line %d: %s\n", __LINE__, hipGetErrorName(err_code));
		return -1;
	}

	
    	// initial path chosen with a greedy heuristic, stored in current_path
    	int current_path[NUM_CITIES];
    	int best_dist = greedy_path_dist(current_path, 0);
    	
    	printf("Greedy best Distance: %d\n", best_dist);
  	puts("Greedy path: ");
  	
  	for (int i=0; i<NUM_CITIES; ++i)
  	{
  		printf("%d\t", current_path[i]);
  	}
  	printf("\n");
    	
    	err_code = hipMemcpy(memory_block,
    		current_path,
    		sizeof(int) * NUM_CITIES,
    		hipMemcpyHostToDevice
    	);
    	
    	if (err_code)
	{
        	printf("[!] Cuda Error at line %d: %s\n", __LINE__, hipGetErrorName(err_code));
        	return -1;
	}
		
  	err_code = hipMemcpy(memory_block + NUM_CITIES,
  		&best_dist,
  		sizeof(int),
  		hipMemcpyHostToDevice
  	);

	if (err_code)
        {
                printf("[!] Cuda Error at line %d: %s\n", __LINE__, hipGetErrorName(err_code));
                return -1;
        }
        

  	// Start the profiler
    	hipProfilerStart();

  	// Call the control thread
  	cuda_opt2<<<1, 1>>>(memory_block, 0);        
	
	// Wait for the GPU to finish
  	hipDeviceSynchronize();

    	// Stop the profiler
    	hipProfilerStop();
    
  	err_code = hipGetLastError();
  	
  	if (err_code)
        {
                printf("[!] Cuda Error at line %d: %s\n", __LINE__, hipGetErrorName(err_code));
                return -1;
        }
        
        // Copy best distance from GPU into best_dist
  	err_code = hipMemcpy(&best_dist,
  		memory_block + NUM_CITIES,
  		sizeof(int),
  		hipMemcpyDeviceToHost
  	);

	if (err_code)
        {
                printf("[!] Cuda Error at line %d: %s\n", __LINE__, hipGetErrorName(err_code));
                return -1;
        }
        
        // Copy the best path from GPU into current_path
        hipMemcpy(current_path,
  		memory_block,
  		sizeof(int) * (NUM_CITIES),
  		hipMemcpyDeviceToHost
  	);

  	
  	printf("Best Distance: %d\n", best_dist);
  	puts("Path: ");
  	
  	for (int i=0; i<NUM_CITIES; ++i)
  	{
  		printf("%d\t", current_path[i]);
  	}
  	printf("\n");
  	
  	hipFree(memory_block);
  	return 0;
}
