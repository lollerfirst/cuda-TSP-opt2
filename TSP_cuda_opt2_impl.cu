#include "hip/hip_runtime.h"
#include <time.h>
#include <stdlib.h>
#include <stdio.h>
#include <stdbool.h>
#include <stdint.h>
#include <assert.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_fp16.h>
#include <mma.h>

#ifndef GENERATION_SEED
#define GENERATION_SEED 1
#endif

#ifndef NUM_CITIES
#define NUM_CITIES 100
#endif

#define MAX_DISTANCE 3267
#define MEM_ALIGNMENT 32
#define BLOCK_SIZE 1024
#define WARP_SIZE 32
#define STRIDE 16

#define BUFFER_LEN ((NUM_CITIES * (NUM_CITIES - 1)) / 2)
#define NUM_OPTS (((NUM_CITIES * (NUM_CITIES - 3)) / 2) + 1)

#define ALIGN(__X) ((((__X) / MEM_ALIGNMENT) + 1) * MEM_ALIGNMENT)

// Distance matrix represented in compressed form
static __half cities[BUFFER_LEN];

// indexes the compressed sparse matrix holding the distances
__inline__ __host__ __device__ size_t triu_index(const int i, const int j)
{
	const int side_i = NUM_CITIES - (i+1);
	const int side_j = NUM_CITIES - (j+1);
	const int sub_area_i = side_i*(side_i+1)/2;
	const int sub_area_j = side_j*(side_j+1)/2;

	return (((BUFFER_LEN - sub_area_i) + j - i - 1) * (i < j)) +
		(((BUFFER_LEN - sub_area_j) + i - j - 1) * (j < i));
}

// find greatest swap_a such that f(swap_a) <= index, with f(n) = n*(n+1)/2
// find swap_b as f(swap_a) - index - 1
__inline__ __device__ void calculate_swap_indices(int* swap_b, int* swap_a, const int index)
{
	*swap_a = __float2int_rd((1.0f + sqrtf(static_cast<float>(1+8*index))) / 2.0f);
	*swap_b = ((swap_a[0] * (swap_a[0] + 1)) / 2) - index;
	++(*swap_a);
}

// build the data structure on the host
void build_cities(unsigned int seed)
{
	srand(seed);
	 
	int i;
	for (i=0; i<BUFFER_LEN; ++i)
	{	

		cities[i] = __int2half_rn(rand() % MAX_DISTANCE);
	}
}

float greedy_path_dist(int* path, int initial_idx)
{
	float distance = 0.0f;
	int i;
	
	bool visited_cities[NUM_CITIES] = {0};
	
	path[0] = initial_idx;
	visited_cities[initial_idx] = true;
	
	
	// For every node in the path
	for (i=1; i<NUM_CITIES+1; ++i)
	{		
		if (i != NUM_CITIES)
		{
			float best_dist = MAX_DISTANCE+1;
			int best_idx = 0;
			int j;
			
			// For every possible link
			for (j=0; j<NUM_CITIES; ++j)
			{
				float local_distance = __half2float(cities[triu_index(path[i-1], j)]);

				if (path[i-1] != j &&
					local_distance <= best_dist &&
					!visited_cities[j])
				{
					best_dist = local_distance;
					best_idx = j;
				}
			}
			
			visited_cities[best_idx] = true;
			path[i] = best_idx;
		}
		else
		{
			path[i] = initial_idx;
		}

		distance += __half2float(cities[triu_index(path[i-1], path[i])]);
	}
	
	return distance;
}

__inline__ __device__ bool trylock(int* mutex)
{
	// Aquire the lock with an atomic compare exchange
	int old = atomicCAS(mutex, 0, 1);
	
	if (old == 1)
	{
		return false;
	}
	else
	{
		return true;
	}
}

__inline__ __device__ void unlock(int* mutex)
{
	// Release the lock with an atomic exchange
	(void) atomicExch(mutex, 0);
}

struct __align__(32) SharedMem
{
	half arr1[BLOCK_SIZE * STRIDE];
	half arr2[BLOCK_SIZE * STRIDE];
	int lock;
};

__device__ __inline__ void load_matrix_a(half* A, half* device_cities, half* cached_values)
{
	#pragma unroll
	for (int i=0; i<4; ++i)
	{
		A[threadIdx.x * STRIDE + i] = cached_values[i];
	}

    // convert to negatives the first 4 values
    long* tmp = reinterpret_cast<long*>(cached_values);
    *tmp |= 0x8000800080008000;

	#pragma unroll
	for (int i=4; i<8; ++i)
	{
		A[threadIdx.x * STRIDE + i] = cached_values[i];
	}
}

__device__ __inline__ void load_matrix_b(half* B, half truth_1, half truth_2)
{
	B[threadIdx.x * STRIDE] = 0x3C00;
	B[threadIdx.x * STRIDE + 1] = truth_1;
	B[threadIdx.x * STRIDE + 2] = truth_2;
	B[threadIdx.x * STRIDE + 3] = 0x3C00;
	B[threadIdx.x * STRIDE + 4] = 0x3C00;
	B[threadIdx.x * STRIDE + 5] = truth_1;
	B[threadIdx.x * STRIDE + 6] = truth_2;
	B[threadIdx.x * STRIDE + 7] = 0x3C00;

	#pragma unroll
	for (int i=8; i<STRIDE; ++i)
	{
		B[threadIdx.x * STRIDE + i] = 0x0000;
	}
}

// Worker threads
__global__ void cuda_calculate_opts(
	half* device_cities,
	int* memory,
	int initial_idx)
{

	const int aligned_unit = ALIGN(sizeof(int) * 2 + sizeof(float)) / sizeof(int);
	const int start_unit = ALIGN(sizeof(int) * (NUM_CITIES+1) + sizeof(float)) / sizeof(int);

	// Thread identification
	int tid = threadIdx.x + (blockIdx.x * blockDim.x);
	
	// Boundaries control
	if (tid >= NUM_OPTS)
	{
		return;
	}


	__shared__ SharedMem block_mem;

	half* A = block_mem.arr1;
	half* B = block_mem.arr2;
	// RESULT MATRIX C IS A + B IN MEMORY TERMS
	float* C = reinterpret_cast<float*>(block_mem.arr1);
	int& lock = block_mem.lock;
	
	half cached_values[8];
	int swap_a, swap_b;
	half truth_1, truth_2;
	float distance;

	// Fragments  
	nvcuda::wmma::fragment<nvcuda::wmma::matrix_a, 16, 16, 16, half, nvcuda::wmma::row_major> a_frag;
	nvcuda::wmma::fragment<nvcuda::wmma::matrix_b, 16, 16, 16, half, nvcuda::wmma::row_major> b_frag;
	nvcuda::wmma::fragment<nvcuda::wmma::accumulator, 16, 16, 16, float> c_frag;

	// Get pointers to current path and current distance
	int* current_path = memory;
	float* f_current_distance = reinterpret_cast<float*>(current_path) + NUM_CITIES+1;
	
	// Get pointers to output indices and output distance
	int* output = memory + start_unit + aligned_unit * blockIdx.x;
	float* f_output_distance = reinterpret_cast<float*>(output) + 2;
	
	// Calculate the swap indices for this thread:
	calculate_swap_indices(&swap_b, &swap_a, tid);

	distance = *f_current_distance;
	truth_1 = __int2half_rn(swap_b + 1 != swap_a);
	truth_2 = __int2half_rn(swap_a - 1 != swap_b);
	
	// Cache distances because we reload A and B multiple times
	cached_values[0] = device_cities[triu_index(current_path[swap_b-1], current_path[swap_b])];
	cached_values[1] = device_cities[triu_index(current_path[swap_b], current_path[swap_b+1])];
	cached_values[2] = device_cities[triu_index(current_path[swap_a-1], current_path[swap_a])];
	cached_values[3] = device_cities[triu_index(current_path[swap_a], current_path[swap_a+1])];
	cached_values[4] = device_cities[triu_index(current_path[swap_b-1], current_path[swap_a])];
	cached_values[5] = device_cities[triu_index(current_path[swap_a], current_path[swap_b+1])];
	cached_values[6] = device_cities[triu_index(current_path[swap_a-1], current_path[swap_b])];
	cached_values[7] = device_cities[triu_index(current_path[swap_b], current_path[swap_a+1])];
	
	load_matrix_a(A, device_cities, cached_values);
	load_matrix_b(B, truth_1, truth_2);
	
	// Load tensor core registers: first half of warp
	nvcuda::wmma::load_matrix_sync(a_frag, A + ((threadIdx.x & 0xFFFFFFE0) * STRIDE), STRIDE);
	nvcuda::wmma::load_matrix_sync(b_frag, B + ((threadIdx.x & 0xFFFFFFE0) * STRIDE), STRIDE);
	nvcuda::wmma::fill_fragment(c_frag, 0.0f);

	// Perform fused multiply add
	nvcuda::wmma::mma_sync(c_frag, a_frag, b_frag, c_frag);
	
	// Store result into C
	nvcuda::wmma::store_matrix_sync(C + ((threadIdx.x & 0xFFFFFFE0) * STRIDE), c_frag, STRIDE, nvcuda::wmma::mem_row_major);

	// Retrieve corresponding accumulated value, which is on the diagonal of the matrix
	((threadIdx.x % 32) < 16) ? distance += C[threadIdx.x * STRIDE + threadIdx.x % 16] : 0;
	
	// Reload A and B
	load_matrix_a(A, device_cities, cached_values);
	load_matrix_b(B, truth_1, truth_2);

	// Load tensor core registers: second half of warp.
	nvcuda::wmma::load_matrix_sync(a_frag, A + (((threadIdx.x & 0xFFFFFFE0) + 16) * STRIDE), STRIDE);
	nvcuda::wmma::load_matrix_sync(b_frag, B + (((threadIdx.x & 0xFFFFFFE0) + 16) * STRIDE), STRIDE);
	nvcuda::wmma::fill_fragment(c_frag, 0.0f);

	// Perform fused multiply add
	nvcuda::wmma::mma_sync(c_frag, a_frag, b_frag, c_frag);

	// Store result into B
	nvcuda::wmma::store_matrix_sync(C + ((threadIdx.x & 0xFFFFFFE0) * STRIDE), c_frag, STRIDE, nvcuda::wmma::mem_row_major);

	// Each thread reads his own cell of the diagonal of B
	((threadIdx.x % 32) >= 16) ? distance += C[threadIdx.x * STRIDE + threadIdx.x % 16] : 0;

	// initialize calculated distance with 0 ~~ default value/not improved
	*f_output_distance = 0.0f;

	// Block-wide sync
	__syncthreads();

	// Acquire the lock
	while (trylock(&lock) == false);
	
	if (distance < (*f_current_distance) &&
		((*f_output_distance) == 0.0f || distance < (*f_output_distance)))
	{
		*f_output_distance = distance;
	
		output[0] = swap_b;
		output[1] = swap_a;
	}	
	
	// Release the lock
	unlock(&lock);
	__syncwarp();
	
	return;
}

// Control thread
__global__ void cuda_opt2(__half* device_cities, int* memory_block, int initial_idx)
{

	const int num_blocks = (NUM_OPTS + BLOCK_SIZE - 1) / BLOCK_SIZE;
	const int aligned_unit = ALIGN(sizeof(int) * 2 + sizeof(float)) / sizeof(int);
	const int start_unit = ALIGN(sizeof(int) * (NUM_CITIES+1) + sizeof(float)) / sizeof(int);

	float* f_memory_ptr = reinterpret_cast<float*>(memory_block);

	float new_best_dist = f_memory_ptr[NUM_CITIES + 1];
	float old_best_dist = new_best_dist + 10.0f;
	
	while (new_best_dist < old_best_dist) 
	{

		// save previous calculated distance
		old_best_dist = new_best_dist;

		// Launch kernel that computes paths and distances
		cuda_calculate_opts<<<num_blocks, BLOCK_SIZE>>>(
			device_cities,
			memory_block,
			initial_idx
		);

		// Wait for child grid to terminate
		hipDeviceSynchronize();

		hipError_t err_code = hipPeekAtLastError();
		assert(err_code == hipSuccess);

		// retrieve best calculated distance amongst various blocks
		int best_index = -1;
		for (int i=0; i<num_blocks; ++i)
		{
			float calc_distance = f_memory_ptr[start_unit + aligned_unit * i + 2];
			if (calc_distance > 0.0f && calc_distance < new_best_dist)
			{
				new_best_dist = calc_distance;
				best_index = i; 
			}
		}

		if (best_index != -1)
		{
			// apply the swap
			int& swap_b = memory_block[start_unit + aligned_unit * best_index];
			int& swap_a = memory_block[start_unit + aligned_unit * best_index + 1];
			
			int temp = memory_block[swap_a];
			memory_block[swap_a] = memory_block[swap_b];
			memory_block[swap_b] = temp;

			f_memory_ptr[NUM_CITIES+1] = new_best_dist;
		}
	}

	
	return;
}


int main(void)
{
	__half* device_cities;
	struct timespec begin, end;
	int* memory_block;

	// Errors
	hipError_t err_code;

	// Build the data structure
	build_cities(GENERATION_SEED);

	if (err_code)
	{
		printf("[!] Cuda Error at line %d: %s\n", __LINE__, hipGetErrorName(err_code));
		return -1;
	}

	// Allocate device cities
	err_code = hipMalloc(&device_cities, BUFFER_LEN * sizeof(__half));

	if (err_code)
	{
		printf("[!] Cuda Error at line %d: %s\n", __LINE__, hipGetErrorName(err_code));
		return -1;
	}

	// Copy cities from host to device, cities is costant
	err_code = hipMemcpy(device_cities,
		cities,
		sizeof(__half) * (BUFFER_LEN),
		hipMemcpyHostToDevice
	);

	if (err_code)
	{
		printf("[!] Cuda Error at line %d: %s\n", __LINE__, hipGetErrorName(err_code));
		return -1;
	}

	// Calculate number of blocks necessary
	const int num_blocks = (NUM_OPTS + BLOCK_SIZE - 1) / BLOCK_SIZE;

	
	// Calculate memory block size:
	// current_path + float_distance + (2 * swap_indices + float_distance) * number_of_blocks
	const size_t memory_block_size = ALIGN(sizeof(int) * (NUM_CITIES+1) + sizeof(float)) + 
		ALIGN(sizeof(int) * 2 + sizeof(float)) * num_blocks;

	// Allocate memory
	hipMalloc((void**) &memory_block,
	    memory_block_size
	);
	
	if (err_code)
	{
		printf("[!] Cuda Error at line %d: %s\n", __LINE__, hipGetErrorName(err_code));
		return -1;
	}


	// initial path chosen with a greedy heuristic, stored in current_path
	int current_path[NUM_CITIES+1];
	float best_dist = greedy_path_dist(current_path, 0);
	
	printf("Greedy best Distance: %f\n", best_dist);
  	puts("Greedy path: ");
  	
  	for (int i=0; i<NUM_CITIES+1; ++i)
  	{
  		printf("%d\t", current_path[i]);
  	}
  	printf("\n");
    	
	err_code = hipMemcpy(memory_block,
		current_path,
		sizeof(int) * (NUM_CITIES+1),
		hipMemcpyHostToDevice
	);
    	
    if (err_code)
	{
		printf("[!] Cuda Error at line %d: %s\n", __LINE__, hipGetErrorName(err_code));
		return -1;
	}
		
  	err_code = hipMemcpy(memory_block + (NUM_CITIES+1),
  		&best_dist,
  		sizeof(float),
  		hipMemcpyHostToDevice
  	);

	if (err_code)
	{
		printf("[!] Cuda Error at line %d: %s\n", __LINE__, hipGetErrorName(err_code));
		return -1;
	}
        

	clock_gettime(CLOCK_MONOTONIC_RAW, &begin);

  	// Call the control thread
  	cuda_opt2<<<1, 1>>>(device_cities, memory_block, 0);        
	
	// Wait for the GPU to finish
  	hipDeviceSynchronize();

	clock_gettime(CLOCK_MONOTONIC_RAW, &end);


  	err_code = hipGetLastError();
  	
  	if (err_code)
	{
		printf("[!] Cuda Error at line %d: %s\n", __LINE__, hipGetErrorName(err_code));
		return -1;
	}
        

	fprintf (stderr, "Total GPU time = %.9f seconds\n",
            (end.tv_nsec - begin.tv_nsec) / 1000000000.0 +
            (end.tv_sec  - begin.tv_sec));

    // Copy best distance from GPU into best_dist
  	err_code = hipMemcpy(&best_dist,
  		memory_block + (NUM_CITIES+1),
  		sizeof(float),
  		hipMemcpyDeviceToHost
  	);

	if (err_code)
	{
		printf("[!] Cuda Error at line %d: %s\n", __LINE__, hipGetErrorName(err_code));
		return -1;
	}
        
	// Copy the best path from GPU into current_path
	hipMemcpy(current_path,
		memory_block,
		sizeof(int) * (NUM_CITIES+1),
		hipMemcpyDeviceToHost
  	);

  	
  	printf("Best Distance: %f\n", best_dist);
  	puts("Path: ");
  	
  	for (int i=0; i<NUM_CITIES+1; ++i)
  	{
  		printf("%d\t", current_path[i]);
  	}
  	printf("\n");
  	#include <time.h>
#include <stdlib.h>
#include <stdio.h>
#include <stdbool.h>
#include <stdint.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_fp16.h>
#include <mma.h>

#ifndef GENERATION_SEED
#define GENERATION_SEED 1
#endif

#ifndef NUM_CITIES
#define NUM_CITIES 100
#endif

#define MAX_DISTANCE 3267
#define MEM_ALIGNMENT 32
#define BLOCK_SIZE 1024
#define WARP_SIZE 32
#define STRIDE 16

#define BUFFER_LEN ((NUM_CITIES * (NUM_CITIES - 1)) / 2)
#define NUM_OPTS (((NUM_CITIES * (NUM_CITIES - 3)) / 2) + 1)

#define ALIGN(__X) ((((__X) / MEM_ALIGNMENT) + 1) * MEM_ALIGNMENT)

// Distance matrix represented in compressed form
static __half cities[BUFFER_LEN];

// indexes the compressed sparse matrix holding the distances
__inline__ __host__ __device__ size_t triu_index(const int i, const int j)
{
	const int side_i = NUM_CITIES - (i+1);
	const int side_j = NUM_CITIES - (j+1);
	const int sub_area_i = side_i*(side_i+1)/2;
	const int sub_area_j = side_j*(side_j+1)/2;

	return (((BUFFER_LEN - sub_area_i) + j - i - 1) * (i < j)) +
		(((BUFFER_LEN - sub_area_j) + i - j - 1) * (j < i));
}

// find greatest swap_a such that f(swap_a) <= index, with f(n) = n*(n+1)/2
// find swap_b as f(swap_a) - index - 1
__inline__ __device__ void calculate_swap_indices(int* swap_b, int* swap_a, const int index)
{
	*swap_a = __float2int_rd((1.0f + sqrtf(static_cast<float>(1+8*index))) / 2.0f);
	*swap_b = ((swap_a[0] * (swap_a[0] + 1)) / 2) - index;
	++(*swap_a);
}

// build the data structure on the host
void build_cities(unsigned int seed)
{
	srand(seed);
	 
	int i;
	for (i=0; i<BUFFER_LEN; ++i)
	{	

		cities[i] = __int2half_rn(rand() % MAX_DISTANCE);
	}
}

float greedy_path_dist(int* path, int initial_idx)
{
	float distance = 0.0f;
	int i;
	
	bool visited_cities[NUM_CITIES] = {0};
	
	path[0] = initial_idx;
	visited_cities[initial_idx] = true;
	
	
	// For every node in the path
	for (i=1; i<NUM_CITIES+1; ++i)
	{		
		if (i != NUM_CITIES)
		{
			float best_dist = MAX_DISTANCE+1;
			int best_idx = 0;
			int j;
			
			// For every possible link
			for (j=0; j<NUM_CITIES; ++j)
			{
				float local_distance = __half2float(cities[triu_index(path[i-1], j)]);

				if (path[i-1] != j &&
					local_distance <= best_dist &&
					!visited_cities[j])
				{
					best_dist = local_distance;
					best_idx = j;
				}
			}
			
			visited_cities[best_idx] = true;
			path[i] = best_idx;
		}
		else
		{
			path[i] = initial_idx;
		}

		distance += __half2float(cities[triu_index(path[i-1], path[i])]);
	}
	
	return distance;
}

__inline__ __device__ bool trylock(int* mutex)
{
	// Aquire the lock with an atomic compare exchange
	int old = atomicCAS(mutex, 0, 1);
	
	if (old == 1)
	{
		return false;
	}
	else
	{
		return true;
	}
}

__inline__ __device__ void unlock(int* mutex)
{
	// Release the lock with an atomic exchange
	(void) atomicExch(mutex, 0);
}

struct __align__(32) SharedMem
{
	half arr1[BLOCK_SIZE * STRIDE];
	half arr2[BLOCK_SIZE * STRIDE];
	int lock;
};

__device__ __inline__ void load_matrix_a(half* A, half* device_cities, half* cached_values)
{
	#pragma unroll
	for (int i=0; i<4; ++i)
	{
		A[threadIdx.x * STRIDE + i] = cached_values[i];
	}

    // convert to negatives the first 4 values
    long* tmp = reinterpret_cast<long*>(cached_values);
    *tmp |= 0x8000800080008000;

	#pragma unroll
	for (int i=4; i<8; ++i)
	{
		A[threadIdx.x * STRIDE + i] = cached_values[i];
	}
}

__device__ __inline__ void load_matrix_b(half* B, half truth_1, half truth_2)
{
	B[threadIdx.x * STRIDE] = 0x3C00;
	B[threadIdx.x * STRIDE + 1] = truth_1;
	B[threadIdx.x * STRIDE + 2] = truth_2;
	B[threadIdx.x * STRIDE + 3] = 0x3C00;
	B[threadIdx.x * STRIDE + 4] = 0x3C00;
	B[threadIdx.x * STRIDE + 5] = truth_1;
	B[threadIdx.x * STRIDE + 6] = truth_2;
	B[threadIdx.x * STRIDE + 7] = 0x3C00;

	#pragma unroll
	for (int i=8; i<STRIDE; ++i)
	{
		B[threadIdx.x * STRIDE + i] = 0x0000;
	}
}

// Worker threads
__global__ void cuda_calculate_opts(
	half* device_cities,
	int* memory,
	int initial_idx)
{

	const int aligned_unit = ALIGN(sizeof(int) * 2 + sizeof(float)) / sizeof(int);
	const int start_unit = ALIGN(sizeof(int) * (NUM_CITIES+1) + sizeof(float)) / sizeof(int);

	// Thread identification
	int tid = threadIdx.x + (blockIdx.x * blockDim.x);
	
	// Boundaries control
	if (tid >= NUM_OPTS)
	{
		return;
	}


	__shared__ SharedMem block_mem;

	half* A = block_mem.arr1;
	half* B = block_mem.arr2;
	// RESULT MATRIX C IS A + B IN MEMORY TERMS
	float* C = reinterpret_cast<float*>(block_mem.arr1);
	int& lock = block_mem.lock;
	
	half cached_values[8];
	int swap_a, swap_b;
	half truth_1, truth_2;
	float distance;

	// Fragments  
	nvcuda::wmma::fragment<nvcuda::wmma::matrix_a, 16, 16, 16, half, nvcuda::wmma::row_major> a_frag;
	nvcuda::wmma::fragment<nvcuda::wmma::matrix_b, 16, 16, 16, half, nvcuda::wmma::row_major> b_frag;
	nvcuda::wmma::fragment<nvcuda::wmma::accumulator, 16, 16, 16, float> c_frag;

	// Get pointers to current path and current distance
	int* current_path = memory;
	float* f_current_distance = reinterpret_cast<float*>(current_path) + NUM_CITIES+1;
	
	// Get pointers to output indices and output distance
	int* output = memory + start_unit + aligned_unit * blockIdx.x;
	float* f_output_distance = reinterpret_cast<float*>(output) + 2;
	
	// Calculate the swap indices for this thread:
	calculate_swap_indices(&swap_b, &swap_a, tid);

	distance = *f_current_distance;
	truth_1 = __int2half_rn(swap_b + 1 != swap_a);
	truth_2 = __int2half_rn(swap_a - 1 != swap_b);
	
	// Cache distances because we reload A and B multiple times
	cached_values[0] = device_cities[triu_index(current_path[swap_b-1], current_path[swap_b])];
	cached_values[1] = device_cities[triu_index(current_path[swap_b], current_path[swap_b+1])];
	cached_values[2] = device_cities[triu_index(current_path[swap_a-1], current_path[swap_a])];
	cached_values[3] = device_cities[triu_index(current_path[swap_a], current_path[swap_a+1])];
	cached_values[4] = device_cities[triu_index(current_path[swap_b-1], current_path[swap_a])];
	cached_values[5] = device_cities[triu_index(current_path[swap_a], current_path[swap_b+1])];
	cached_values[6] = device_cities[triu_index(current_path[swap_a-1], current_path[swap_b])];
	cached_values[7] = device_cities[triu_index(current_path[swap_b], current_path[swap_a+1])];
	
	load_matrix_a(A, device_cities, cached_values);
	load_matrix_b(B, truth_1, truth_2);
	
	// Load tensor core registers: first half of warp
	nvcuda::wmma::load_matrix_sync(a_frag, A + ((threadIdx.x & 0xFFFFFFE0) * STRIDE), STRIDE);
	nvcuda::wmma::load_matrix_sync(b_frag, B + ((threadIdx.x & 0xFFFFFFE0) * STRIDE), STRIDE);
	nvcuda::wmma::fill_fragment(c_frag, 0.0f);

	// Perform fused multiply add
	nvcuda::wmma::mma_sync(c_frag, a_frag, b_frag, c_frag);
	
	// Store result into C
	nvcuda::wmma::store_matrix_sync(C + ((threadIdx.x & 0xFFFFFFE0) * STRIDE), c_frag, STRIDE, nvcuda::wmma::mem_row_major);

	// Retrieve corresponding accumulated value, which is on the diagonal of the matrix
	((threadIdx.x % 32) < 16) ? distance += C[threadIdx.x * STRIDE + threadIdx.x % 16] : 0;
	
	// Reload A and B
	load_matrix_a(A, device_cities, cached_values);
	load_matrix_b(B, truth_1, truth_2);

	// Load tensor core registers: second half of warp.
	nvcuda::wmma::load_matrix_sync(a_frag, A + (((threadIdx.x & 0xFFFFFFE0) + 16) * STRIDE), STRIDE);
	nvcuda::wmma::load_matrix_sync(b_frag, B + (((threadIdx.x & 0xFFFFFFE0) + 16) * STRIDE), STRIDE);
	nvcuda::wmma::fill_fragment(c_frag, 0.0f);

	// Perform fused multiply add
	nvcuda::wmma::mma_sync(c_frag, a_frag, b_frag, c_frag);

	// Store result into B
	nvcuda::wmma::store_matrix_sync(C + ((threadIdx.x & 0xFFFFFFE0) * STRIDE), c_frag, STRIDE, nvcuda::wmma::mem_row_major);

	// Each thread reads his own cell of the diagonal of B
	((threadIdx.x % 32) >= 16) ? distance += C[threadIdx.x * STRIDE + threadIdx.x % 16] : 0;

	// initialize calculated distance with 0 ~~ default value/not improved
	*f_output_distance = 0.0f;

	// Block-wide sync
	__syncthreads();

	// Acquire the lock
	while (trylock(&lock) == false);
	
	if (distance < (*f_current_distance) &&
		((*f_output_distance) == 0.0f || distance < (*f_output_distance)))
	{
		*f_output_distance = distance;
	
		output[0] = swap_b;
		output[1] = swap_a;
	}	
	
	// Release the lock
	unlock(&lock);
	__syncwarp();
	
	return;
}

// Control thread
__global__ void cuda_opt2(__half* device_cities, int* memory_block, int initial_idx)
{

	const int num_blocks = (NUM_OPTS + BLOCK_SIZE - 1) / BLOCK_SIZE;
	const int aligned_unit = ALIGN(sizeof(int) * 2 + sizeof(float)) / sizeof(int);
	const int start_unit = ALIGN(sizeof(int) * (NUM_CITIES+1) + sizeof(float)) / sizeof(int);

	float* f_memory_ptr = reinterpret_cast<float*>(memory_block);

	float new_best_dist = f_memory_ptr[NUM_CITIES + 1];
	float old_best_dist = new_best_dist + 10.0f;
	
	while (new_best_dist < old_best_dist) 
	{

		// save previous calculated distance
		old_best_dist = new_best_dist;

		// Launch kernel that computes paths and distances
		cuda_calculate_opts<<<num_blocks, BLOCK_SIZE>>>(
			device_cities,
			memory_block,
			initial_idx
		);

		// Wait for child grid to terminate
		hipDeviceSynchronize();

		hipError_t err_code = hipPeekAtLastError();
		assert(err_code == hipSuccess);

		// retrieve best calculated distance amongst various blocks
		int best_index = -1;
		for (int i=0; i<num_blocks; ++i)
		{
			float calc_distance = f_memory_ptr[start_unit + aligned_unit * i + 2];
			if (calc_distance > 0.0f && calc_distance < new_best_dist)
			{
				new_best_dist = calc_distance;
				best_index = i; 
			}
		}

		if (best_index != -1)
		{
			// apply the swap
			int& swap_b = memory_block[start_unit + aligned_unit * best_index];
			int& swap_a = memory_block[start_unit + aligned_unit * best_index + 1];
			
			int temp = memory_block[swap_a];
			memory_block[swap_a] = memory_block[swap_b];
			memory_block[swap_b] = temp;

			f_memory_ptr[NUM_CITIES+1] = new_best_dist;
		}
	}

	
	return;
}


int main(void)
{
	__half* device_cities;
	struct timespec begin, end;
	int* memory_block;

	// Errors
	hipError_t err_code;

	// Build the data structure
	build_cities(GENERATION_SEED);

	// Allocate device cities
	err_code = hipMalloc(&device_cities, BUFFER_LEN * sizeof(__half));

	if (err_code)
	{
		printf("[!] Cuda Error at line %d: %s\n", __LINE__, hipGetErrorName(err_code));
		return -1;
	}

	// Copy cities from host to device, cities is costant
	err_code = hipMemcpy(device_cities,
		cities,
		sizeof(__half) * (BUFFER_LEN),
		hipMemcpyHostToDevice
	);

	if (err_code)
	{
		printf("[!] Cuda Error at line %d: %s\n", __LINE__, hipGetErrorName(err_code));
		return -1;
	}

	// Calculate number of blocks necessary
	const int num_blocks = (NUM_OPTS + BLOCK_SIZE - 1) / BLOCK_SIZE;

	
	// Calculate memory block size:
	// current_path + float_distance + (2 * swap_indices + float_distance) * number_of_blocks
	const size_t memory_block_size = ALIGN(sizeof(int) * (NUM_CITIES+1) + sizeof(float)) + 
		ALIGN(sizeof(int) * 2 + sizeof(float)) * num_blocks;

	// Allocate memory
	hipMalloc((void**) &memory_block,
	    memory_block_size
	);
	
	if (err_code)
	{
		printf("[!] Cuda Error at line %d: %s\n", __LINE__, hipGetErrorName(err_code));
		return -1;
	}


	// initial path chosen with a greedy heuristic, stored in current_path
	int current_path[NUM_CITIES+1];
	float best_dist = greedy_path_dist(current_path, 0);
	
	printf("Greedy best Distance: %f\n", best_dist);
  	puts("Greedy path: ");
  	
  	for (int i=0; i<NUM_CITIES+1; ++i)
  	{
  		printf("%d\t", current_path[i]);
  	}
  	printf("\n");
    	
	err_code = hipMemcpy(memory_block,
		current_path,
		sizeof(int) * (NUM_CITIES+1),
		hipMemcpyHostToDevice
	);
    	
    if (err_code)
	{
		printf("[!] Cuda Error at line %d: %s\n", __LINE__, hipGetErrorName(err_code));
		return -1;
	}
		
  	err_code = hipMemcpy(memory_block + (NUM_CITIES+1),
  		&best_dist,
  		sizeof(float),
  		hipMemcpyHostToDevice
  	);

	if (err_code)
	{
		printf("[!] Cuda Error at line %d: %s\n", __LINE__, hipGetErrorName(err_code));
		return -1;
	}
        

	clock_gettime(CLOCK_MONOTONIC_RAW, &begin);

  	// Call the control thread
  	cuda_opt2<<<1, 1>>>(device_cities, memory_block, 0);        
	
	// Wait for the GPU to finish
  	hipDeviceSynchronize();

	clock_gettime(CLOCK_MONOTONIC_RAW, &end);


  	err_code = hipGetLastError();
  	
  	if (err_code)
	{
		printf("[!] Cuda Error at line %d: %s\n", __LINE__, hipGetErrorName(err_code));
		return -1;
	}
        

	fprintf (stderr, "Total GPU time = %.9f seconds\n",
            (end.tv_nsec - begin.tv_nsec) / 1000000000.0 +
            (end.tv_sec  - begin.tv_sec));

    // Copy best distance from GPU into best_dist
  	err_code = hipMemcpy(&best_dist,
  		memory_block + (NUM_CITIES+1),
  		sizeof(float),
  		hipMemcpyDeviceToHost
  	);

	if (err_code)
	{
		printf("[!] Cuda Error at line %d: %s\n", __LINE__, hipGetErrorName(err_code));
		return -1;
	}
        
	// Copy the best path from GPU into current_path
	hipMemcpy(current_path,
		memory_block,
		sizeof(int) * (NUM_CITIES+1),
		hipMemcpyDeviceToHost
  	);

  	
  	printf("Best Distance: %f\n", best_dist);
  	puts("Path: ");
  	
  	for (int i=0; i<NUM_CITIES+1; ++i)
  	{
  		printf("%d\t", current_path[i]);
  	}
  	printf("\n");
  	
  	hipFree(memory_block);
  	return 0;
}